#include "hip/hip_runtime.h"
#define MIN(x, y) (((x) < (y)) ? (x) : (y))

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
/*****************************************************************/
static double tti = 0;
__global__ void fillSudoku(char* memory,int* stats)
{
  int i,current_poss,j,temp,mat_i,mat_j,k;
  char* block_memory = memory+(81*blockIdx.x);

  __shared__ int row_used_numbers[9];
  __shared__ int col_used_numbers[9];
  __shared__ int cell_used_numbers[9];
  __shared__ char progress_flag;
  __shared__ char done_flag;
  __shared__ char error_flag;
  __shared__ int min_forks;
  __shared__ int scheduling_thread;

    // check whether all blocks are idle or not.This shouldn't happen
    if (blockIdx.x==0){ //first block
      if (threadIdx.x==0) {
        progress_flag=0;
      }
      __syncthreads();
      for(i=threadIdx.x;i<gridDim.x;i+=blockDim.x){
        if (stats[i] > 0){
          progress_flag=1;
        }
      }
      __syncthreads();
      if (progress_flag==0 and threadIdx.x == 0){
        //no active block terminate.
        if (threadIdx.x==0) printf("no active blocks...terminating\n");
        stats[gridDim.x]=2;
      }
    }

    //If block is active work on it.
    if(stats[blockIdx.x]==1){
      if (threadIdx.x==0) {
        error_flag = 0;  //set to 1 if the board is wrongs
        done_flag = 0;  //set to 1 if the board is solved
        progress_flag=1; //set to 0 if no reterministic progress can be made.
      }
      __syncthreads();

      while(!error_flag && !done_flag &&progress_flag ){
        __syncthreads();
        // 1st check whether the board is valid and fill X_used_numbers arrays for rows,columns and cells.
        //*************************
        if (threadIdx.x<9){
          // TODO optimize here such that there wraps does row/col/cell.
            row_used_numbers[threadIdx.x] = 0;
            col_used_numbers[threadIdx.x] = 0;
            cell_used_numbers[threadIdx.x] = 0;
            for(i=0;i<9;i++){
                //rows
                temp = block_memory[threadIdx.x*9+i];
                if (temp) { //!=0
                    if ((row_used_numbers[threadIdx.x]>>(temp-1)) & 1){
                        // This is bad, you have the same number in the same row. This solution fails
                        error_flag=10+i;
                    }
                    //set n'th bit to 1.
                    row_used_numbers[threadIdx.x] |= 1<<(temp-1);
                }
                //columns
                temp = block_memory[i*9+threadIdx.x];
                if (temp) { //!=0
                    if ((col_used_numbers[threadIdx.x]>>(temp-1)) & 1){
                        // This is bad, you have the same number in the same column. This solution fails
                        error_flag=20+i;
                    }
                    //set n'th bit to 1.
                    col_used_numbers[threadIdx.x] |= 1<<(temp-1);
                }
            }
            //cells
            for (i=(threadIdx.x/3)*3;i<((threadIdx.x/3+1)*3);i++){
              for (j=(threadIdx.x%3)*3;j<((threadIdx.x%3+1)*3);j++){
                temp = block_memory[i*9+j];
                if (temp) { //!=0
                    if ((cell_used_numbers[threadIdx.x]>>(temp-1)) & 1){
                        // This is bad, you have the same number in the same cell. This solution fails
                        error_flag=30+i;
                    }
                    //set n'th bit to 1.
                    cell_used_numbers[threadIdx.x] |= 1<<(temp-1);
                }
              }
            }

            }
        __syncthreads();
        if (error_flag==0){
            if (threadIdx.x==0) {
              progress_flag = 0;
              done_flag = 1;
            }
            __syncthreads();
           if (threadIdx.x<81){
                // 2nd for each cell calculate available numbers(row_used OR col_used OR cell_used) and if there is one 0
                //*************************
                current_poss = 0;
                mat_i = threadIdx.x/9;
                mat_j = threadIdx.x%9;
                if (block_memory[threadIdx.x] == 0){
                    done_flag = 0;
                    current_poss = (row_used_numbers[mat_i] | col_used_numbers[mat_j] | cell_used_numbers[(mat_i/3)*3+(mat_j/3)]);
                    //printf("thredix=%d,current_poss=%d\n",threadIdx.x,current_poss);
                    temp = 0; // temp for count
                    for (i=0;i<9;i++){
                      if ((current_poss & (1<<i))==0){
                        if (temp){ //if there is a zero found already
                          temp = 10;
                          break;
                        }
                        else{
                          temp = i+1;
                        }
                      }
                    }
                    if (temp==0){
                        
                        error_flag = 1;
                        progress_flag = 1;
                    }
                    else if (temp<=9){
                    
                      block_memory[threadIdx.x] = temp;
                      progress_flag = 1;
                    }
                }
            }
          }
          __syncthreads();
        }
        __syncthreads();
        if (done_flag) {
          if (threadIdx.x==0){
            memcpy(memory+gridDim.x*81,block_memory,81);
            stats[gridDim.x]=2;
          }
        }
        else if (error_flag!=0){
          if (threadIdx.x==0)
            stats[blockIdx.x]=0;
        }

        else if (progress_flag==0) {
          // Implement scheduling here
          if (threadIdx.x==0){
            min_forks = 9;
            scheduling_thread = blockDim.x; //bigger then and threadID
          }
          __syncthreads();
          //lets find out what is the minimum number of possible digits for any cell.
          temp = 0; // temp for count
          if(current_poss!=0){
            for (i=0;i<9;i++){
              if ((current_poss & (1<<i))==0){
                  temp++;
                }
            }
            atomicMin(&min_forks,temp);
          }
          __syncthreads();
          //then out of most ambigous cells lets choose the one with smallest threadIdx.
          if (temp==min_forks){
            atomicMin(&scheduling_thread,threadIdx.x);
          }
          __syncthreads();
          if (scheduling_thread==threadIdx.x){
            //Find a suitable block to schedule the fork for each extra value.
            k = 1;
            j=0; //to continue wherever we stayed on the previous search.
            for (i=0;i<9;i++){
              if ((current_poss & (1<<i))==0){
                    if (k==1) {
                      // first possibility stays with the current block
                      block_memory[threadIdx.x] = i+1;
                    }
                    else{
                      // look for suitable block
                      for (;j<gridDim.x;j++) {
                        atomicCAS(stats+j,0,gridDim.x*blockIdx.x+threadIdx.x+2); //unique identifier>1
                        if (stats[j] == (gridDim.x*blockIdx.x+threadIdx.x+2)){
                          //succesful scheduling
                     
                          memcpy(memory+j*81,block_memory,81);
                          memory[j*81+threadIdx.x] = i+1;
                          stats[j] = 1;
                          break;
                        }
                      }
                    }
                    k++;
                }
            }
          }
          __syncthreads();
        }
    }
  }
__global__ void controller(char* arr_dev,int* block_stat,int nBlocks, int nThreads){
  int  i = 0;
  while (block_stat[nBlocks]!=2 && i<55){//stats[gridDim.x]==2 means, solution is coppied to the last 81 char of memory.
    fillSudoku<<<nBlocks,nThreads>>>(arr_dev,block_stat);
    hipDeviceSynchronize();
    i++;
  }
}

void  gpu_sudoku_solver(char* arr,int version)
{
  char *arr_dev;
  int *block_stat;

  int nThreads = 96; // wrap_size 32, each thread will have responsible from one cell.
  int nBlocks = 20000; // max available concurent blocks/searches running.
  int memSize = 81*(nBlocks+1); // 0.81 MB for N=9
  //copy array and create a new one temp. last block/stat is for the result
  hipMalloc((void**) &block_stat,(nBlocks+1)*sizeof(int));
  hipMemset(block_stat, 0, (nBlocks+1)*sizeof(int));
  hipMemset(block_stat, 1, 1);
  hipMalloc((void**) &arr_dev,memSize);
  hipMemcpy(arr_dev,arr,81,hipMemcpyHostToDevice);

  if (version==1){
    printf("Block=%d,threads=%d starting\n",nBlocks,nThreads);

    clock_t start1 = clock();
    controller<<<1,1>>>(arr_dev,block_stat,nBlocks,nThreads);
    clock_t end1 = clock();
    double cpu_time_used = ((double) (end1 - start1)) / CLOCKS_PER_SEC;
    tti = tti + cpu_time_used;
    printf("%f\t%f\n",cpu_time_used,tti);
    hipError_t err = hipGetLastError();
      if (err != hipSuccess)
          printf("Error: %s\n", hipGetErrorString(err));
  }
  else{
    printf("Invalid version'\n");
    exit(1);
  }
  hipMemcpy(arr,arr_dev+81*nBlocks,81,hipMemcpyDeviceToHost);
  hipFree(arr_dev);
  hipFree(block_stat);
}

void readSudoku(FILE *fp, char *sudoku){
  int i,j;
  for (i=0;i<9;i++){
    fscanf(fp, "%s", sudoku);
    for (j=0;j<9;j++){
      sudoku[j] = sudoku[j] - 48; //48 = char '0'
    }
    sudoku = sudoku + 9;
  }
}
int main(int argc, char *argv[])
{
  hipFree(0);
  hipSetDevice ( 1 );
  int version=1,flag=1,i;
  char cur_sudoku[81];
  char out_name[200];
  double time_taken;
  clock_t start, end;
  FILE *inp_fp,*out_fp;

  if (argc <=3 && argc>1  )
    {
      memset(out_name, '\0', sizeof(out_name));
      strcpy(out_name, argv[1]);
      i = 0 ;
      while (out_name[i] != '\0' && out_name[i] != '.'){
        i ++;
      }
      if (out_name[i] == '\0'){
        fprintf(stderr, "The input file should be like X.in: %s!\n",out_name[i]);
        exit(1);
      }
      else{
        out_name[i+1] = 's';
        out_name[i+2] = 'o';
        out_name[i+3] = 'l';
        out_name[i+4] = '\0';
      }
      out_fp = fopen(out_name,"w");
      if (out_fp == NULL) {
        fprintf(stderr, "Can't open output file %s!\n",out_name);
        exit(1);
      }

      inp_fp = fopen(argv[1],"r");
      if (inp_fp == NULL) {
        fprintf(stderr, "Can't open input file %s!\n",argv[1]);
        exit(1);
      }
    }

    while (flag!=-1){
      readSudoku(inp_fp,cur_sudoku);
      gpu_sudoku_solver(cur_sudoku,version);
      fgetc( inp_fp );
      flag=fgetc( inp_fp );
    }
    fclose(inp_fp);
    fclose(out_fp);

}
